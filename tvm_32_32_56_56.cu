//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[2];
    __shared__ float pad_temp_shared[384];
    __shared__ float kernel_shared[4608];
    float pad_temp_shared_local[8];
    float kernel_shared_local[4];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
            pad_temp_shared[(((((((int)threadIdx.z) * 12) + (((int)threadIdx.y) * 6)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3)) < 57)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 57)) ? data[(((((((((rc_outer * 50176) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) - 57))] : 0.000000e+00f);
        }
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 36; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
            kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((int)threadIdx.z) * 288) + (rc_outer * 144)) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 36)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
        }
        __syncthreads();
        for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
            for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
                for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
                    for (int ax1 = 0; ax1 < 4; ++ax1) {
                        pad_temp_shared_local[(ax1)] = pad_temp_shared[(((((((rc_inner_outer * 96) + (ax1 * 24)) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + ((int)threadIdx.x)) + rx_inner_outer))];
                        pad_temp_shared_local[((ax1 + 4))] = pad_temp_shared[((((((((rc_inner_outer * 96) + (ax1 * 24)) + (((int)threadIdx.y) * 6)) + (ry_inner_outer * 6)) + ((int)threadIdx.x)) + rx_inner_outer) + 2))];
                    }
                    for (int ax11 = 0; ax11 < 4; ++ax11) {
                        kernel_shared_local[(ax11)] = kernel_shared[((((((((int)threadIdx.z) * 144) + (rc_inner_outer * 36)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer))];
                    }
                    for (int rc_inner_inner = 0; rc_inner_inner < 4; ++rc_inner_inner) {
                        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[(rc_inner_inner)]));
                        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[((rc_inner_inner + 4))] * kernel_shared_local[(rc_inner_inner)]));
                    }
                }
            }
        }
    }
    compute[((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = compute_local[(0)];
    compute[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)) + 2))] = compute_local[(1)];
}

int main(int argc, char *argv[]){
    dim3 grid(14,28,1);
    dim3 block(2,2,32);

    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}