//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[4];
    __shared__ float pad_temp_shared[784];
    __shared__ float kernel_shared[256];
    float pad_temp_shared_local[4];
    float kernel_shared_local[16];
#pragma unroll
    for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
        compute_local[(ff_c_init)] = 0.000000e+00f;
    }
    for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
        for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
#pragma unroll
            for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
                __syncthreads();
#pragma unroll
                for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
                    pad_temp_shared[(((((((int)threadIdx.z) * 196) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (ry_outer + (((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 7)) % 7))) && ((ry_outer + (((((int)threadIdx.y) * 4) + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 7)) % 7)) < 8)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 7)))) && ((rx_outer + (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 7)) < 8)) ? data[((((((((rc_outer * 784) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 28)) + (ry_outer * 7)) + ((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)) + rx_outer) - 8))] : 0.000000e+00f);
                }
#pragma unroll
                for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
                    if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 10) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4)) < 16) {
                        if (((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 10)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 256) {
                            if ((((((int)threadIdx.y) * 10) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 64) {
                                if (((((int)threadIdx.x) * 2) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 10) {
                                    kernel_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 10)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 9216) + (((int)threadIdx.z) * 2304)) + (((((((int)threadIdx.y) * 10) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >> 4) * 576)) + (rc_outer * 144)) + (((((((int)threadIdx.y) * 10) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) & 15) * 9)) + (ry_outer * 3)) + rx_outer))];
                                }
                            }
                        }
                    }
                }
                __syncthreads();
#pragma unroll
                for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
#pragma unroll
                    for (int ax1 = 0; ax1 < 4; ++ax1) {
                        pad_temp_shared_local[(ax1)] = pad_temp_shared[(((((rc_inner_outer * 196) + (ax1 * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
                    }
#pragma unroll
                    for (int ax0 = 0; ax0 < 4; ++ax0) {
#pragma unroll
                        for (int ax11 = 0; ax11 < 4; ++ax11) {
                            kernel_shared_local[(((ax0 * 4) + ax11))] = kernel_shared[(((((((int)threadIdx.z) * 64) + (ax0 * 16)) + (rc_inner_outer * 4)) + ax11))];
                        }
                    }
#pragma unroll
                    for (int rc_inner_inner = 0; rc_inner_inner < 4; ++rc_inner_inner) {
#pragma unroll
                        for (int ff_c = 0; ff_c < 4; ++ff_c) {
                            compute_local[(ff_c)] = (compute_local[(ff_c)] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[(((ff_c * 4) + rc_inner_inner))]));
                        }
                    }
                }
            }
        }
    }
#pragma unroll
    for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
        compute[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (ff_inner_inner_inner * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(ff_inner_inner_inner)];
    }
}

//grid=(1,1,2),  block=(7,7,4)
int main(int argc, char *argv[]){
    dim3 grid(1,1,2);
    dim3 block(7,7,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}