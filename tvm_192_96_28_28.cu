//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[32];
    __shared__ float pad_temp_shared[2016];
    __shared__ float kernel_shared[1152];
    float pad_temp_shared_local[12];
    float kernel_shared_local[96];
    for (int ff_c_init = 0; ff_c_init < 8; ++ff_c_init) {
        compute_local[(ff_c_init)] = 0.000000e+00f;
        compute_local[((ff_c_init + 16))] = 0.000000e+00f;
        compute_local[((ff_c_init + 8))] = 0.000000e+00f;
        compute_local[((ff_c_init + 24))] = 0.000000e+00f;
    }
    for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
        for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
                pad_temp_shared[(((((((int)threadIdx.z) * 1008) + (((int)threadIdx.y) * 504)) + (((int)threadIdx.x) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 168) / 28))) && (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 168) / 28)) < 29)) && (1 <= (rx_outer + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 28)))) && ((rx_outer + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 28)) < 29)) ? data[(((((((((rc_outer * 9408) + (((int)threadIdx.z) * 4704)) + (((int)threadIdx.y) * 2352)) + ((((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 168) * 784)) + (((int)blockIdx.y) * 112)) + rx_outer) + (((((int)threadIdx.x) * 18) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 168)) - 29))] : 0.000000e+00f);
            }
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 11; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
                if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 36)) < 32) {
                    if ((((((int)threadIdx.z) * 192) + (((int)threadIdx.y) * 96)) + (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 3)) < 384) {
                        if (((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 11)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1152) {
                            if ((((((int)threadIdx.y) * 288) + (((int)threadIdx.x) * 11)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 576) {
                                if (((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
                                    kernel_shared[(((((((int)threadIdx.z) * 576) + (((int)threadIdx.y) * 288)) + (((int)threadIdx.x) * 11)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 27648)) + (((int)threadIdx.y) * 13824)) + ((((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 36) * 1728)) + (rc_outer * 108)) + ((((((int)threadIdx.x) * 11) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 36) * 3)) + rx_outer))];
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
            for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
                for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
                    for (int ax1 = 0; ax1 < 6; ++ax1) {
                        pad_temp_shared_local[(ax1)] = pad_temp_shared[((((((rc_inner_outer * 1008) + (ax1 * 168)) + (((int)threadIdx.y) * 28)) + (ry_inner_outer * 28)) + ((int)threadIdx.x)))];
                        pad_temp_shared_local[((ax1 + 6))] = pad_temp_shared[(((((((rc_inner_outer * 1008) + (ax1 * 168)) + (((int)threadIdx.y) * 28)) + (ry_inner_outer * 28)) + ((int)threadIdx.x)) + 56))];
                    }
                    for (int ax0 = 0; ax0 < 8; ++ax0) {
                        for (int ax11 = 0; ax11 < 6; ++ax11) {
                            kernel_shared_local[(((ax0 * 6) + ax11))] = kernel_shared[((((((((int)threadIdx.z) * 288) + (ax0 * 36)) + (rc_inner_outer * 18)) + (ax11 * 3)) + ry_inner_outer))];
                            kernel_shared_local[((((ax0 * 6) + ax11) + 48))] = kernel_shared[(((((((((int)threadIdx.z) * 288) + (ax0 * 36)) + (rc_inner_outer * 18)) + (ax11 * 3)) + ry_inner_outer) + 576))];
                        }
                    }
                    for (int rc_inner_inner = 0; rc_inner_inner < 6; ++rc_inner_inner) {
                        for (int ff_c = 0; ff_c < 8; ++ff_c) {
                            compute_local[(ff_c)] = (compute_local[(ff_c)] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[(((ff_c * 6) + rc_inner_inner))]));
                            compute_local[((ff_c + 16))] = (compute_local[((ff_c + 16))] + (pad_temp_shared_local[(rc_inner_inner)] * kernel_shared_local[((((ff_c * 6) + rc_inner_inner) + 48))]));
                            compute_local[((ff_c + 8))] = (compute_local[((ff_c + 8))] + (pad_temp_shared_local[((rc_inner_inner + 6))] * kernel_shared_local[(((ff_c * 6) + rc_inner_inner))]));
                            compute_local[((ff_c + 24))] = (compute_local[((ff_c + 24))] + (pad_temp_shared_local[((rc_inner_inner + 6))] * kernel_shared_local[((((ff_c * 6) + rc_inner_inner) + 48))]));
                        }
                    }
                }
            }
        }
    }
    for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 8; ++ff_inner_inner_inner) {
        compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)))] = compute_local[(ff_inner_inner_inner)];
        compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 12544))] = compute_local[((ff_inner_inner_inner + 16))];
        compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 56))] = compute_local[((ff_inner_inner_inner + 8))];
        compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 6272)) + (ff_inner_inner_inner * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)) + 12600))] = compute_local[((ff_inner_inner_inner + 24))];
    }
}

//grid=(1,7,3),  block=(28,2,2)
int main(int argc, char *argv[]){
    dim3 grid(1,7,3);
    dim3 block(28,2,2);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}