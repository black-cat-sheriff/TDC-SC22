//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[4];
    __shared__ float pad_temp_shared[168];
    __shared__ float kernel_shared[6];
    float pad_temp_shared_local[6];
    float kernel_shared_local[6];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(2)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    compute_local[(3)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 64; ++rc_outer) {
        __syncthreads();
        pad_temp_shared[(((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)))] = (((1 <= ((int)threadIdx.y)) && (1 <= ((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)))) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) - 29))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 1))] = ((1 <= ((int)threadIdx.y)) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) - 28))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 2))] = (((1 <= ((int)threadIdx.y)) && (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)) < 27)) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) - 27))] : 0.000000e+00f);
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 6) {
            if (((int)threadIdx.x) < 1) {
                kernel_shared[((((int)threadIdx.x) + ((int)threadIdx.y)))] = kernel[(((((((int)blockIdx.z) * 1152) + (((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) * 576)) + (rc_outer * 9)) + ((((int)threadIdx.x) + ((int)threadIdx.y)) % 3)))];
            }
        }
        __syncthreads();
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 1))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 3))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 4))];
        kernel_shared_local[(0)] = kernel_shared[(0)];
        kernel_shared_local[(3)] = kernel_shared[(3)];
        kernel_shared_local[(1)] = kernel_shared[(1)];
        kernel_shared_local[(4)] = kernel_shared[(4)];
        kernel_shared_local[(2)] = kernel_shared[(2)];
        kernel_shared_local[(5)] = kernel_shared[(5)];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        __syncthreads();
        pad_temp_shared[(((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)))] = ((1 <= ((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3))) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) - 1))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 1))] = data[(((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)))];
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 2))] = ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)) < 27) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) + 1))] : 0.000000e+00f);
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 6) {
            if (((int)threadIdx.x) < 1) {
                kernel_shared[((((int)threadIdx.x) + ((int)threadIdx.y)))] = kernel[((((((((int)blockIdx.z) * 1152) + (((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) * 576)) + (rc_outer * 9)) + ((((int)threadIdx.x) + ((int)threadIdx.y)) % 3)) + 3))];
            }
        }
        __syncthreads();
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 1))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 3))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 4))];
        kernel_shared_local[(0)] = kernel_shared[(0)];
        kernel_shared_local[(3)] = kernel_shared[(3)];
        kernel_shared_local[(1)] = kernel_shared[(1)];
        kernel_shared_local[(4)] = kernel_shared[(4)];
        kernel_shared_local[(2)] = kernel_shared[(2)];
        kernel_shared_local[(5)] = kernel_shared[(5)];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        __syncthreads();
        pad_temp_shared[(((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)))] = (((((int)threadIdx.y) < 27) && (1 <= ((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)))) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) + 27))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 1))] = ((((int)threadIdx.y) < 27) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) + 28))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.y) * 6) + (((int)threadIdx.x) * 3)) + 2))] = (((((int)threadIdx.y) < 27) && (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) * 3)) < 27)) ? data[((((((rc_outer * 784) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) * 3)) + 29))] : 0.000000e+00f);
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 6) {
            if (((int)threadIdx.x) < 1) {
                kernel_shared[((((int)threadIdx.x) + ((int)threadIdx.y)))] = kernel[((((((((int)blockIdx.z) * 1152) + (((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) * 576)) + (rc_outer * 9)) + ((((int)threadIdx.x) + ((int)threadIdx.y)) % 3)) + 6))];
            }
        }
        __syncthreads();
        pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 6) + ((int)threadIdx.x)))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 1))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 3))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 2))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 6) + ((int)threadIdx.x)) + 4))];
        kernel_shared_local[(0)] = kernel_shared[(0)];
        kernel_shared_local[(3)] = kernel_shared[(3)];
        kernel_shared_local[(1)] = kernel_shared[(1)];
        kernel_shared_local[(4)] = kernel_shared[(4)];
        kernel_shared_local[(2)] = kernel_shared[(2)];
        kernel_shared_local[(5)] = kernel_shared[(5)];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    }
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = compute_local[(0)];
    compute[((((((((int)blockIdx.z) * 1568) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)) + 784))] = compute_local[(2)];
    compute[((((((((int)blockIdx.z) * 1568) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)) + 2))] = compute_local[(1)];
    compute[((((((((int)blockIdx.z) * 1568) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)) + 786))] = compute_local[(3)];
}


//grid=(7,1,16),  block=(2,28,1)
int main(int argc, char *argv[]){
    dim3 grid(7,1,16);
    dim3 block(2,28,1);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}