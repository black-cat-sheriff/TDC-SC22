//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[2];
    __shared__ float pad_temp_shared[108];
    __shared__ float kernel_shared[144];
    float pad_temp_shared_local[12];
    float kernel_shared_local[24];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 24; ++rc_outer) {
        __syncthreads();
        pad_temp_shared[(((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)))] = (((((1 <= ((((((int)threadIdx.x) * 8) % 27) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 8) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 8) % 9))) && (((((int)threadIdx.x) * 8) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + (((((int)threadIdx.x) * 8) / 27) * 49)) + ((((((int)threadIdx.x) * 8) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 8) % 9)) - 8))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 1) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 1) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 1) % 9))) && ((((((int)threadIdx.x) * 8) + 1) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 1) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 1) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 1) % 9)) - 8))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 2) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 2) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 2) % 9))) && ((((((int)threadIdx.x) * 8) + 2) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 2) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 2) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 2) % 9)) - 8))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 3) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 3) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 3) % 9))) && ((((((int)threadIdx.x) * 8) + 3) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 3) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 3) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 3) % 9)) - 8))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 4) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 4) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 4) % 9))) && ((((((int)threadIdx.x) * 8) + 4) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 4) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 4) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 4) % 9)) - 8))] : 0.000000e+00f);
        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 5) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 5) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 5) % 9))) && ((((((int)threadIdx.x) * 8) + 5) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 5) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 5) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 5) % 9)) - 8))] : 0.000000e+00f);
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 8) + 6) / 27)) < 4) {
            if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 6) / 9)) < 12) {
                if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 102) {
                    if (((int)threadIdx.x) < 6) {
                        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 6) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 6) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 6) % 9))) && ((((((int)threadIdx.x) * 8) + 6) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 6) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 6) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 6) % 9)) - 8))] : 0.000000e+00f);
                    }
                }
            }
        }
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 8) + 7) / 27)) < 4) {
            if (((((int)threadIdx.z) * 6) + (((((int)threadIdx.x) * 8) + 7) / 9)) < 12) {
                if (((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) < 101) {
                    if (((int)threadIdx.x) < 6) {
                        pad_temp_shared[((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= (((((((int)threadIdx.x) * 8) + 7) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 8) + 7) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 8) + 7) % 9))) && ((((((int)threadIdx.x) * 8) + 7) % 9) < 8)) ? data[((((((((rc_outer * 196) + (((int)threadIdx.z) * 98)) + ((((((int)threadIdx.x) * 8) + 7) / 27) * 49)) + (((((((int)threadIdx.x) * 8) + 7) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 8) + 7) % 9)) - 8))] : 0.000000e+00f);
                    }
                }
            }
        }
        kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + (((((int)threadIdx.x) * 11) / 36) * 864)) + (rc_outer * 36)) + ((((int)threadIdx.x) * 11) % 36)))];
        kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 1) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 1) % 36)))];
        kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 2) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 2) % 36)))];
        kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 3) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 3) % 36)))];
        kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 4) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 4) % 36)))];
        kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 5) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 5) % 36)))];
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 11) + 6) / 36)) < 4) {
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 16) {
                if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 46) {
                    if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 138) {
                        if (((int)threadIdx.x) < 6) {
                            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 6) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 6) % 36)))];
                        }
                    }
                }
            }
        }
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 11) + 7) / 36)) < 4) {
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 16) {
                if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 48) {
                    if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 137) {
                        if (((int)threadIdx.x) < 6) {
                            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 7) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 7) % 36)))];
                        }
                    }
                }
            }
        }
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 11) + 8) / 36)) < 4) {
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 16) {
                if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 48) {
                    if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 136) {
                        if (((int)threadIdx.x) < 6) {
                            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 8) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 8) % 36)))];
                        }
                    }
                }
            }
        }
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 11) + 9) / 36)) < 4) {
            if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 15) {
                if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 45) {
                    if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 135) {
                        if (((int)threadIdx.x) < 6) {
                            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 9) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 9) % 36)))];
                        }
                    }
                }
            }
        }
        if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 11) + 10) / 36)) < 4) {
            if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 16) {
                if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 48) {
                    if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 134) {
                        if (((int)threadIdx.x) < 6) {
                            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 1728)) + ((((((int)threadIdx.x) * 11) + 10) / 36) * 864)) + (rc_outer * 36)) + (((((int)threadIdx.x) * 11) + 10) % 36)))];
                        }
                    }
                }
            }
        }
        __syncthreads();
        pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
        pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
        pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
        pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
        pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
        pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
        pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
        kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
        kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
        kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
        kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
        kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
        kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
        kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
        kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
        kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
        kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
        kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
        kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
        kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
        kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
        kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
        kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
        kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
        kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
        kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
        kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
        kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
        kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
        kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
        kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(16)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(17)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(18)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(19)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(20)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
        pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
        pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
        pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
        pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
        pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
        pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
        kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
        kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
        kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
        kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
        kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
        kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
        kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
        kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
        kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
        kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
        kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
        kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
        kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
        kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
        kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
        kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
        kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
        kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
        kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
        kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
        kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
        kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
        kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(16)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(17)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(18)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(19)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(20)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
        pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
        pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
        pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
        pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
        pad_temp_shared_local[(4)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
        pad_temp_shared_local[(5)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
        pad_temp_shared_local[(6)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
        pad_temp_shared_local[(7)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
        pad_temp_shared_local[(8)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
        pad_temp_shared_local[(9)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
        pad_temp_shared_local[(10)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
        pad_temp_shared_local[(11)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
        kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
        kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
        kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
        kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
        kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
        kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
        kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
        kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
        kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
        kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
        kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
        kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
        kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
        kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
        kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
        kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
        kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
        kernel_shared_local[(18)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
        kernel_shared_local[(19)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
        kernel_shared_local[(20)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
        kernel_shared_local[(21)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
        kernel_shared_local[(22)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
        kernel_shared_local[(23)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(15)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(16)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(17)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(18)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(19)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(20)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(21)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(22)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(23)]));
    }
    compute[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 98)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
    compute[((((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 98)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)) + 49))] = compute_local[(1)];
}

//grid=(1,7,16),  block=(7,1,2)
int main(int argc, char *argv[]){
    dim3 grid(1,7,16);
    dim3 block(7,1,2);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}