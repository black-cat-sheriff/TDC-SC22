//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[16];
    __shared__ float pad_temp_shared[2592];
    __shared__ float kernel_shared[576];
    float pad_temp_shared_local[24];
    float kernel_shared_local[12];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(8)] = 0.000000e+00f;
    compute_local[(4)] = 0.000000e+00f;
    compute_local[(12)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    compute_local[(9)] = 0.000000e+00f;
    compute_local[(5)] = 0.000000e+00f;
    compute_local[(13)] = 0.000000e+00f;
    compute_local[(2)] = 0.000000e+00f;
    compute_local[(10)] = 0.000000e+00f;
    compute_local[(6)] = 0.000000e+00f;
    compute_local[(14)] = 0.000000e+00f;
    compute_local[(3)] = 0.000000e+00f;
    compute_local[(11)] = 0.000000e+00f;
    compute_local[(7)] = 0.000000e+00f;
    compute_local[(15)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
        __syncthreads();
        pad_temp_shared[((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)))] = (((((1 <= ((((int)blockIdx.y) * 16) + ((((int)threadIdx.x) * 21) / 18))) && (((((int)blockIdx.y) * 16) + ((((int)threadIdx.x) * 21) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) * 21) % 18)))) && (((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) * 21) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + (((((int)threadIdx.x) * 21) / 18) * 112)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) * 21) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 1))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 1) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 1) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 1) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 1) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 1) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 1) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 2))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 2) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 2) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 2) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 2) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 2) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 2) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 3))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 3) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 3) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 3) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 3) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 3) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 3) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 4))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 4) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 4) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 4) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 4) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 4) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 4) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 5))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 5) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 5) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 5) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 5) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 5) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 5) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 6))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 6) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 6) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 6) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 6) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 6) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 6) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 7))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 7) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 7) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 7) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 7) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 7) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 7) % 18)) - 113))] : 0.000000e+00f);
        pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 8))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 8) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 8) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 8) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 8) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 8) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 8) % 18)) - 113))] : 0.000000e+00f);
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 9) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 9) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2583) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 639) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 9))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 9) / 18))) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 9) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 9) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 9) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 9) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 10) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 10) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2582) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 638) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 10))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 10) / 18))) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 10) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 10) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 10) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 10) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 11) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 11) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2581) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 637) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 11))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 11) / 18))) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 11) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 11) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 11) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 11) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 12) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 12) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2580) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 636) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 12))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 12) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 12) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 12) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 12) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 12) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 12) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 13) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 13) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2579) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 635) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 13))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 13) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 13) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 13) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 13) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 13) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 13) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 14) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 14) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2578) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 634) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 14))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 14) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 14) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 14) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 14) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 14) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 14) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 15) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 15) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2577) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 633) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 15))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 15) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 15) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 15) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 15) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 15) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 15) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 16) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 16) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2576) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 632) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 16))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 16) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 16) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 16) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 16) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 16) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 16) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 17) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 17) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2575) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 631) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 17))] = (((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 17) / 18))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 17) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 17) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 17) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 17) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 17) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 18) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + ((((int)threadIdx.x) * 21) / 18)) < 143) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2574) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 630) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 18))] = ((((((((int)blockIdx.y) * 16) + ((((int)threadIdx.x) * 21) / 18)) < 112) && (1 <= ((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) * 21) % 18)))) && (((((int)blockIdx.x) * 16) + ((((int)threadIdx.x) * 21) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + (((((int)threadIdx.x) * 21) / 18) * 112)) + (((int)blockIdx.x) * 16)) + ((((int)threadIdx.x) * 21) % 18)) - 1))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 19) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 19) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2573) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 629) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 19))] = ((((((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 19) / 18)) < 113) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 1) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 1) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 19) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 1) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 21) + 20) / 324)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 18)) + (((((int)threadIdx.x) * 21) + 20) / 18)) < 144) {
                if ((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) < 2572) {
                    if (((((int)threadIdx.y) * 324) + (((int)threadIdx.x) * 21)) < 628) {
                        if (((int)threadIdx.x) < 15) {
                            pad_temp_shared[(((((((int)threadIdx.z) * 648) + (((int)threadIdx.y) * 324)) + (((int)threadIdx.x) * 21)) + 20))] = ((((((((int)blockIdx.y) * 16) + (((((int)threadIdx.x) * 21) + 20) / 18)) < 113) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 2) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 21) + 2) % 18)) < 113)) ? data[(((((((((rc_outer * 100352) + (((int)threadIdx.z) * 25088)) + (((int)threadIdx.y) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.x) * 21) + 20) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 21) + 2) % 18)) - 113))] : 0.000000e+00f);
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + ((((int)threadIdx.x) * 5) / 72)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + ((((int)threadIdx.x) * 5) / 9)) < 64) {
                if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + ((((int)threadIdx.x) * 5) / 3)) < 192) {
                    if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) < 576) {
                        if (((((int)threadIdx.y) * 72) + (((int)threadIdx.x) * 5)) < 144) {
                            if (((int)threadIdx.x) < 15) {
                                kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 72)) + (((int)threadIdx.x) * 5)))];
                            }
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 5) + 1) / 72)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 5) + 1) / 9)) < 64) {
                if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((((int)threadIdx.x) * 5) + 1) / 3)) < 192) {
                    if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) < 575) {
                        if (((((int)threadIdx.y) * 72) + (((int)threadIdx.x) * 5)) < 143) {
                            if (((int)threadIdx.x) < 15) {
                                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) + 1))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 72)) + (((int)threadIdx.x) * 5)) + 1))];
                            }
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 5) + 2) / 72)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 5) + 2) / 9)) < 64) {
                if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((((int)threadIdx.x) * 5) + 2) / 3)) < 192) {
                    if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) < 574) {
                        if (((((int)threadIdx.y) * 72) + (((int)threadIdx.x) * 5)) < 142) {
                            if (((int)threadIdx.x) < 14) {
                                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) + 2))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 72)) + (((int)threadIdx.x) * 5)) + 2))];
                            }
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 5) + 3) / 72)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 5) + 3) / 9)) < 64) {
                if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + ((((int)threadIdx.x) * 5) / 3)) < 191) {
                    if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) < 573) {
                        if (((((int)threadIdx.y) * 72) + (((int)threadIdx.x) * 5)) < 141) {
                            if (((int)threadIdx.x) < 14) {
                                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) + 3))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 72)) + (((int)threadIdx.x) * 5)) + 3))];
                            }
                        }
                    }
                }
            }
        }
        if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 5) + 4) / 72)) + ((int)threadIdx.y)) < 8) {
            if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 5) + 4) / 9)) < 64) {
                if ((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((((int)threadIdx.x) * 5) + 4) / 3)) < 192) {
                    if ((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) < 572) {
                        if (((((int)threadIdx.y) * 72) + (((int)threadIdx.x) * 5)) < 140) {
                            if (((int)threadIdx.x) < 14) {
                                kernel_shared[(((((((int)threadIdx.z) * 144) + (((int)threadIdx.y) * 72)) + (((int)threadIdx.x) * 5)) + 4))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 72)) + (((int)threadIdx.x) * 5)) + 4))];
                            }
                        }
                    }
                }
            }
        }
        __syncthreads();
        for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
            pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)))];
            pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 144))];
            pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 18))];
            pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 162))];
            pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 36))];
            pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 180))];
            pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 54))];
            pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 198))];
            pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 72))];
            pad_temp_shared_local[(16)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 216))];
            pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 90))];
            pad_temp_shared_local[(17)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 234))];
            pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 324))];
            pad_temp_shared_local[(18)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 468))];
            pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 342))];
            pad_temp_shared_local[(19)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 486))];
            pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 360))];
            pad_temp_shared_local[(20)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 504))];
            pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 378))];
            pad_temp_shared_local[(21)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 522))];
            pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 396))];
            pad_temp_shared_local[(22)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 540))];
            pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 414))];
            pad_temp_shared_local[(23)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 558))];
            kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)))];
            kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 288))];
            kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 3))];
            kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 291))];
            kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 6))];
            kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 294))];
            kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 9))];
            kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 297))];
            kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 12))];
            kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 300))];
            kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 15))];
            kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 303))];
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(6)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(7)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(1)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(7)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(1)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(8)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(2)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(8)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(8)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(2)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(8)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(9)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(9)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(3)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(9)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(9)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(3)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(9)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(9)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(10)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(4)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(10)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(4)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(10)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(4)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(10)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(4)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(10)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(10)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(11)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(11)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(5)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(11)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(11)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(5)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(11)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(11)]));
            pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 1))];
            pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 145))];
            pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 19))];
            pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 163))];
            pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 37))];
            pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 181))];
            pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 55))];
            pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 199))];
            pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 73))];
            pad_temp_shared_local[(16)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 217))];
            pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 91))];
            pad_temp_shared_local[(17)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 235))];
            pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 325))];
            pad_temp_shared_local[(18)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 469))];
            pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 343))];
            pad_temp_shared_local[(19)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 487))];
            pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 361))];
            pad_temp_shared_local[(20)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 505))];
            pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 379))];
            pad_temp_shared_local[(21)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 523))];
            pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 397))];
            pad_temp_shared_local[(22)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 541))];
            pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 415))];
            pad_temp_shared_local[(23)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 559))];
            kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 1))];
            kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 289))];
            kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 4))];
            kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 292))];
            kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 7))];
            kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 295))];
            kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 10))];
            kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 298))];
            kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 13))];
            kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 301))];
            kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 16))];
            kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 304))];
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(6)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(7)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(1)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(7)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(1)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(8)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(2)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(8)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(8)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(2)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(8)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(9)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(9)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(3)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(9)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(9)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(3)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(9)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(9)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(10)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(4)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(10)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(4)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(10)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(4)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(10)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(4)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(10)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(10)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(11)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(11)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(5)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(11)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(11)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(5)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(11)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(11)]));
            pad_temp_shared_local[(0)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 2))];
            pad_temp_shared_local[(12)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 146))];
            pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 20))];
            pad_temp_shared_local[(13)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 164))];
            pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 38))];
            pad_temp_shared_local[(14)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 182))];
            pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 56))];
            pad_temp_shared_local[(15)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 200))];
            pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 74))];
            pad_temp_shared_local[(16)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 218))];
            pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 92))];
            pad_temp_shared_local[(17)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 236))];
            pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 326))];
            pad_temp_shared_local[(18)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 470))];
            pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 344))];
            pad_temp_shared_local[(19)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 488))];
            pad_temp_shared_local[(8)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 362))];
            pad_temp_shared_local[(20)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 506))];
            pad_temp_shared_local[(9)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 380))];
            pad_temp_shared_local[(21)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 524))];
            pad_temp_shared_local[(10)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 398))];
            pad_temp_shared_local[(22)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 542))];
            pad_temp_shared_local[(11)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 416))];
            pad_temp_shared_local[(23)] = pad_temp_shared[(((((rc_inner_outer * 648) + (((int)threadIdx.y) * 72)) + ((int)threadIdx.x)) + 560))];
            kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 2))];
            kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 290))];
            kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 5))];
            kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 293))];
            kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 8))];
            kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 296))];
            kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 11))];
            kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 299))];
            kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 14))];
            kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 302))];
            kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 17))];
            kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + 305))];
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(6)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(6)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(6)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(6)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(7)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(1)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(7)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(7)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(1)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(7)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(7)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(8)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(2)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(8)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(2)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(8)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(2)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(8)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(8)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(9)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(9)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(3)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(9)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(3)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(9)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(3)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(9)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(3)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(9)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(9)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(10)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(10)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(4)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(10)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(4)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(10)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(4)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(10)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(4)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(10)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(10)]));
            compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
            compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(11)]));
            compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
            compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(11)]));
            compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(5)]));
            compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(11)]));
            compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(5)]));
            compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(11)]));
            compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(5)]));
            compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(11)]));
            compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(5)]));
            compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(11)]));
            compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
            compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
            compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
            compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(11)]));
        }
    }
    compute[(((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = compute_local[(0)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 50176))] = compute_local[(8)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 896))] = compute_local[(4)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 51072))] = compute_local[(12)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 112))] = compute_local[(1)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 50288))] = compute_local[(9)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 1008))] = compute_local[(5)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 51184))] = compute_local[(13)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 224))] = compute_local[(2)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 50400))] = compute_local[(10)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 1120))] = compute_local[(6)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 51296))] = compute_local[(14)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 336))] = compute_local[(3)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 50512))] = compute_local[(11)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 1232))] = compute_local[(7)];
    compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 51408))] = compute_local[(15)];
}

//grid=(7,7,4),  block=(16,2,4)
int main(int argc, char *argv[]){
    dim3 grid(7,7,4);
    dim3 block(16,2,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}