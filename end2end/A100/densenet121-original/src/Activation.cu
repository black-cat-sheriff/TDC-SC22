#include "../inc/common.h"
/*
 * hipdnnNanPropagation_t : HIPDNN_NOT_PROPAGATE_NAN,HIPDNN_PROPAGATE_NAN
 * hipdnnActivationMode_t:
 * HIPDNN_ACTIVATION_SIGMOID,
    HIPDNN_ACTIVATION_RELU,
    HIPDNN_ACTIVATION_TANH,
    HIPDNN_ACTIVATION_CLIPPED_RELU,
    HIPDNN_ACTIVATION_ELU,
 *
 */
void Activation::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    hipMalloc(&output,B*C*H*W*sizeof(float));
    checkCUDNN(hipdnnCreate(&activationCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&activationOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(activationOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
    checkCUDNN(hipdnnSetActivationDescriptor(activationDesc,HIPDNN_ACTIVATION_RELU,HIPDNN_NOT_PROPAGATE_NAN,0.0f));
}
float * Activation::forward(float *input) {
    checkCUDNN(hipdnnActivationForward(activationCudnn,activationDesc,&alpha,activationInputDescriptor,input,&beta,activationOutputDescriptor,output));
    return output;
}
