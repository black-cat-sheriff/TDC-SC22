//
// Created by lizhi on 4/1/22.
//

#include <hipDNN.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class Conv{
public:
    unsigned int H;
    unsigned int W;
    unsigned int C;
    unsigned int B;
    unsigned int N;
    unsigned int PAD;
    unsigned int hOut;
    unsigned int wOut;
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    hipdnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    hipdnnTensorDescriptor_t convInputDescriptor;
    hipdnnTensorDescriptor_t convOutputDescriptor;
    hipdnnFilterDescriptor_t convKernelDescriptor;
    hipdnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                    unsigned int pad,unsigned int r,unsigned int s,unsigned int stride);
    float *forward(float *input);
};
void Conv::initialize(unsigned int b,unsigned int c,unsigned int h,unsigned int w,unsigned int n,
                      unsigned int pad,unsigned int r,unsigned int s,unsigned int stride){
    this->B = b;
    this->C = c;
    this->H = h;
    this->W = w;
    this->N = n;
    this->hOut = (H+2*pad - r)/stride + 1;
    this->wOut = (W+2*pad - s)/stride + 1;
    hipMalloc(&kernel,sizeof(float)*C*N*r*s);
    hipMalloc(&this->output,sizeof(float)*B*hOut*wOut*N);
    hipdnnCreate(&convCudnn);
    hipdnnCreateTensorDescriptor(&convInputDescriptor);
    hipdnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    hipdnnCreateFilterDescriptor(&convKernelDescriptor);
    hipdnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/r,
            /*kernel_width=*/s);
    hipdnnCreateConvolutionDescriptor(&convDesc);
    hipdnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/pad,
            /*pad_width=*/pad,
            /*vertical_stride=*/stride,
            /*horizontal_stride=*/stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
                                    HIPDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    hipdnnCreateTensorDescriptor(&convOutputDescriptor);
    hipdnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/N,
            /*image_height=*/hOut,
            /*image_width=*/wOut);
    hipdnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = r*s*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,r*s*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * Conv::forward(float *input) {
    hipMemset(output, 0, B*N*hOut*wOut*sizeof(float));
    checkCUDNN(hipdnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
int main(int argc, char *argv[]){
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    int C = atoi(argv[1]);
    int N = atoi(argv[2]);
    int H = atoi(argv[3]);
    int W = atoi(argv[4]);
    Conv conv;
    conv.initialize(1,C,H,W,N,1,3,3,1);
    float *input = new float[C*H*W];
    float *d_input;
    float inference_time = 0.0f;
    float temp_time;
    hipMalloc(&d_input,C*H*W*sizeof(float));
    for(int i=0;i<1024;++i){
        time_t t;
        srand((unsigned) time(&t));
        for(int j =0;j<C*H*W;++j){
            input[j] = rand() % 10;
        }
        hipMemcpy(d_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start)
        float *out = conv.forward(d_input);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<inference_time/1024<<endl;
    return 0;
}