//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[12];
    __shared__ float pad_temp_shared[72];
    __shared__ float kernel_shared[72];
    float pad_temp_shared_local[12];
    float kernel_shared_local[9];
    for (int xx_c_init = 0; xx_c_init < 4; ++xx_c_init) {
        compute_local[(xx_c_init)] = 0.000000e+00f;
        compute_local[((xx_c_init + 4))] = 0.000000e+00f;
        compute_local[((xx_c_init + 8))] = 0.000000e+00f;
    }
    for (int rc_outer = 0; rc_outer < 80; ++rc_outer) {
        for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
                if ((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 72) {
                    if (((((int)threadIdx.y) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 18) {
                        pad_temp_shared[((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 7) + (((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 36) >> 2))) && (((((int)blockIdx.y) * 7) + (((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 36) >> 2)) < 29)) && (1 <= (((((int)blockIdx.x) * 4) + rx_outer) + ((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)))) && ((((((int)blockIdx.x) * 4) + rx_outer) + ((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)) < 29)) ? data[(((((((((rc_outer * 1568) + (((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 36) * 784)) + (((int)blockIdx.y) * 196)) + ((((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 36) >> 2) * 28)) + (((int)blockIdx.x) * 4)) + rx_outer) + ((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)) - 29))] : 0.000000e+00f);
                    }
                }
            }
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
                if (((((int)threadIdx.z) * 3) + (((int)threadIdx.y) >> 1)) < 12) {
                    if (((((int)threadIdx.z) * 6) + ((int)threadIdx.y)) < 24) {
                        if ((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 72) {
                            if (((((int)threadIdx.y) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 18) {
                                if ((((((int)blockIdx.z) * 12) + (((int)threadIdx.z) * 3)) + (((int)threadIdx.y) >> 1)) < 96) {
                                    kernel_shared[((((((int)threadIdx.z) * 18) + (((int)threadIdx.y) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 17280) + (((int)threadIdx.z) * 4320)) + ((((int)threadIdx.y) >> 1) * 1440)) + (rc_outer * 18)) + ((((int)threadIdx.y) & 1) * 9)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 * 3)) + rx_outer))];
                                }
                            }
                        }
                    }
                }
            }
            __syncthreads();
            for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
                for (int ax2 = 0; ax2 < 3; ++ax2) {
                    for (int ax3 = 0; ax3 < 4; ++ax3) {
                        pad_temp_shared_local[(((ax2 * 4) + ax3))] = pad_temp_shared[(((((rc_inner_outer * 36) + (ax2 * 4)) + (((int)threadIdx.y) * 4)) + ax3))];
                    }
                }
                for (int ax21 = 0; ax21 < 3; ++ax21) {
                    kernel_shared_local[(ax21)] = kernel_shared[((((((int)threadIdx.z) * 6) + (rc_inner_outer * 3)) + ax21))];
                    kernel_shared_local[((ax21 + 3))] = kernel_shared[(((((((int)threadIdx.z) * 6) + (rc_inner_outer * 3)) + ax21) + 24))];
                    kernel_shared_local[((ax21 + 6))] = kernel_shared[(((((((int)threadIdx.z) * 6) + (rc_inner_outer * 3)) + ax21) + 48))];
                }
                for (int ry_inner_inner = 0; ry_inner_inner < 3; ++ry_inner_inner) {
                    for (int xx_c = 0; xx_c < 4; ++xx_c) {
                        compute_local[(xx_c)] = (compute_local[(xx_c)] + (pad_temp_shared_local[(((ry_inner_inner * 4) + xx_c))] * kernel_shared_local[(ry_inner_inner)]));
                        compute_local[((xx_c + 4))] = (compute_local[((xx_c + 4))] + (pad_temp_shared_local[(((ry_inner_inner * 4) + xx_c))] * kernel_shared_local[((ry_inner_inner + 3))]));
                        compute_local[((xx_c + 8))] = (compute_local[((xx_c + 8))] + (pad_temp_shared_local[(((ry_inner_inner * 4) + xx_c))] * kernel_shared_local[((ry_inner_inner + 6))]));
                    }
                }
            }
        }
    }
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 4; ++xx_inner_inner_inner) {
        compute[(((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 196)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + xx_inner_inner_inner))] = compute_local[(xx_inner_inner_inner)];
        compute[((((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 196)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + xx_inner_inner_inner) + 3136))] = compute_local[((xx_inner_inner_inner + 4))];
        compute[((((((((((int)blockIdx.z) * 9408) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 196)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 4)) + xx_inner_inner_inner) + 6272))] = compute_local[((xx_inner_inner_inner + 8))];
    }
}
//grid=(7,4,8),  block=(1,7,4)
int main(int argc, char *argv[]){
    dim3 grid(7,4,8);
    dim3 block(1,7,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}