#include "../inc/common.h"
unsigned int *conv_index = new unsigned int[1];
float *conv_time = new float[100];
int main(int argc,char *argv[]){
    conv_index[0] = 0;
    for(int i=0;i<100;++i){
        conv_time[i] = 0.0f;
    }
    float *input = (float *)malloc(224*224*3*sizeof(float));
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2," ", false);
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112," ");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    /*
     * layer 1
     */
    TkShape layer1_basic0_tk0_shape(64,64,32,64);
    TkShape layer1_basic0_tk1_shape(64,32,32,64);
    TkWeight layer1_basic0_tk0_weight(" "," "," ");
    TkWeight layer1_basic0_tk1_weight(" "," "," ");

    TkShape layer1_basic1_tk0_shape(64,32,32,64);
    TkShape layer1_basic1_tk1_shape(64,32,32,64);
    TkWeight layer1_basic1_tk0_weight(" "," "," ");
    TkWeight layer1_basic1_tk1_weight(" "," "," ");

    BasicBlock layer1_basic0(layer1_basic0_tk0_shape,layer1_basic0_tk1_shape,
                             layer1_basic0_tk0_weight,layer1_basic0_tk1_weight,
                             56,56," "," ");
    BasicBlock layer1_basic1(layer1_basic1_tk0_shape,layer1_basic1_tk1_shape,
                             layer1_basic1_tk0_weight,layer1_basic1_tk1_weight,
                             56,56," "," ");
    /*
     * layer 2
     */
    TkShape layer2_basic0_tk0_shape(64,64,64,128);
    TkShape layer2_basic0_tk1_shape(128,96,64,128);
    TkWeight layer2_basic0_tk0_weight(" "," "," ");
    TkWeight layer2_basic0_tk1_weight(" "," "," ");

    TkShape layer2_basic1_tk0_shape(128,96,64,128);
    TkShape layer2_basic1_tk1_shape(128,96,64,128);
    TkWeight layer2_basic1_tk0_weight(" "," "," ");
    TkWeight layer2_basic1_tk1_weight(" "," "," ");

    BasicBlock_Downsample layer2_basic0(layer2_basic0_tk0_shape,layer2_basic0_tk1_shape,
                                        layer2_basic0_tk0_weight,layer2_basic0_tk1_weight,
                                        56,56,128,
                                        " "," "," "," ");
    BasicBlock layer2_basic1(layer2_basic1_tk0_shape,layer2_basic1_tk1_shape,
                             layer2_basic1_tk0_weight,
                             layer2_basic1_tk1_weight,28,28,
                             " "," ");
    /*
     * layer 3
     */
    TkShape layer3_basic0_tk0_shape(128,96,96,256);
    TkShape layer3_basic0_tk1_shape(256,128,96,256);
    TkWeight layer3_basic0_tk0_weight(" "," "," ");
    TkWeight layer3_basic0_tk1_weight(" "," "," ");

    TkShape layer3_basic1_tk0_shape(256,128,96,256);
    TkShape layer3_basic1_tk1_shape(256,128,96,256);
    TkWeight layer3_basic1_tk0_weight(" "," "," ");
    TkWeight layer3_basic1_tk1_weight(" "," "," ");

    BasicBlock_Downsample layer3_basic0(layer3_basic0_tk0_shape,layer3_basic0_tk1_shape,
                                        layer3_basic0_tk0_weight,layer3_basic0_tk1_weight,
                                        28,28,256,
                                        " "," "," "," ");
    BasicBlock layer3_basic1(layer3_basic1_tk0_shape,layer3_basic1_tk1_shape,
                             layer3_basic1_tk0_weight,
                             layer3_basic1_tk1_weight,14,14,
                             " "," ");

    /*
     * layer 4
     */
    TkShape layer4_basic0_tk0_shape(256,160,160,512);
    TkShape layer4_basic0_tk1_shape(512,192,160,512);
    TkWeight layer4_basic0_tk0_weight(" "," "," ");
    TkWeight layer4_basic0_tk1_weight(" "," "," ");

    TkShape layer4_basic1_tk0_shape(512,192,160,512);
    TkShape layer4_basic1_tk1_shape(512,192,160,512);
    TkWeight layer4_basic1_tk0_weight(" "," "," ");
    TkWeight layer4_basic1_tk1_weight(" "," "," ");

    BasicBlock_Downsample layer4_basic0(layer4_basic0_tk0_shape,layer4_basic0_tk1_shape,
                                        layer4_basic0_tk0_weight,layer4_basic0_tk1_weight,
                                        14,14,512,
                                        " "," "," "," ");
    BasicBlock layer4_basic1(layer4_basic1_tk0_shape,layer4_basic1_tk1_shape,
                             layer4_basic1_tk0_weight,
                             layer4_basic1_tk1_weight,7,7,
                             " "," ");

    Pool avg_pool;
    avg_pool.initialize(1,512,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,512,1,1,1000,0,1,1,1,"../../../weights/resnet50/weights/predictions.bin", true);

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    //hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *out;
    out = conv1_conv.forward(dInput);
    out = conv1_bn.forward(out);
    out = conv1_relu.forward(out);
    out = conv1_max_pool.forward(out);
    out = layer1_basic0.forward(out);
    out = layer1_basic1.forward(out);
    out = layer2_basic0.forward(out);
    out = layer2_basic1.forward(out);
    out = layer3_basic0.forward(out);
    out = layer3_basic1.forward(out);
    out = layer4_basic0.forward(out);
    out = layer4_basic1.forward(out);
    out = predict.forward(out);
    hipDeviceSynchronize();
    cout<<"network construction finished"<<endl;
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        conv_time[i] = 0.0f;
    }
    for(int i=0;i<1024;++i){
        conv_index[0] = 0;
        hipEventRecord(event_start);
        out = conv1_conv.forward(dInput);
        out = conv1_bn.forward(out);
        out = conv1_relu.forward(out);
        out = conv1_max_pool.forward(out);
        out = layer1_basic0.forward(out);
        out = layer1_basic1.forward(out);
        out = layer2_basic0.forward(out);
        out = layer2_basic1.forward(out);
        out = layer3_basic0.forward(out);
        out = layer3_basic1.forward(out);
        out = layer4_basic0.forward(out);
        out = layer4_basic1.forward(out);
        out = predict.forward(out);
        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    for(int i=0;i<100;++i){
        cout<<conv_time[i]/1024<<endl;
    }
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,out,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("resnet18-tk-cudnn.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}