//
// Created by lizhi on 4/2/22.
//
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
__global__ void transform(float *x, float *y, int C,int H,int W){
    for(unsigned int id = threadIdx.x + blockIdx.x*blockDim.x;id<C*H*W;id+=blockDim.x*gridDim.x){
        if(id >= C*H*W){
            return ;
        }
        unsigned int c = id / (H*W);
        unsigned int hw =id % (H*W);
        float v = x[c*H*W+hw];
        y[hw*C+c] = v;
    }
}
int main(int argc, char *argv[]) {
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    float *x;
    hipMalloc(&x, C * H * W * sizeof(float));
    float *y;
    hipMalloc(&y, C * H * W * sizeof(float));
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipEventRecord(event_start);
    transform<<<68, 1024>>>(x, y, C, H, W);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&inference_time, event_start, event_stop);
    cout << inference_time << endl;
    return 0;
}


