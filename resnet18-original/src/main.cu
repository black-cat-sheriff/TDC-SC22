#include "../inc/common.h"
int main(int argc,char *argv[]){
    //float *input = (float *)malloc(224*224*3*sizeof(float));
    //string imagePath = argv[1];
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,2," ", false);
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112," ");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    /*
     * layer 1
     */
    BasicBlock layer1_basic0(64,64,64,56,56," "," "," "," ");
    BasicBlock layer1_basic1(64,64,64,56,56," "," "," "," ");
    /*
     * layer 2
     */
    BasicBlock_Downsample layer2_basic0(64,128,128,128,56,56," "," "," "," "," "," ");
    BasicBlock layer2_basic1(128,128,128,28,28," "," "," "," ");
    /*
     * layer 3
     */
    BasicBlock_Downsample layer3_basic0(128,256,256,256,28,28," "," "," "," "," "," ");
    BasicBlock layer3_basic1(256,256,256,14,14," "," "," "," ");

    /*
     * layer 4
     */
    BasicBlock_Downsample layer4_basic0(256,512,512,512,14,14," "," "," "," "," "," ");
    BasicBlock layer4_basic1(512,512,512,7,7," "," "," "," ");

    Pool avg_pool;
    avg_pool.initialize(1,512,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,512,1,1,1000,0,1,1,1,"../../../weights/resnet50/weights/predictions.bin", true);

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    //hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *out;
    out = conv1_conv.forward(dInput);
    out = conv1_bn.forward(out);
    out = conv1_relu.forward(out);
    out = conv1_max_pool.forward(out);
    out = layer1_basic0.forward(out);
    out = layer1_basic1.forward(out);
    out = layer2_basic0.forward(out);
    out = layer2_basic1.forward(out);
    out = layer3_basic0.forward(out);
    out = layer3_basic1.forward(out);
    out = layer4_basic0.forward(out);
    out = layer4_basic1.forward(out);
    out = predict.forward(out);
    chkerr(hipDeviceSynchronize());
    hipDeviceSynchronize();
    cout<<"network construction finished"<<endl;
    float inference_time = 0.0f;
    for(int i=0;i<1024;++i){
        //string image = imagePath + "/" + to_string(i) + ".bin";
        //load_input(image, 3 * 224 * 224, input);
        //hipMemcpy(dInput,input,224*224*3*sizeof(float),hipMemcpyHostToDevice);
        hipEventRecord(event_start);
        out = conv1_conv.forward(dInput);
        out = conv1_bn.forward(out);
        out = conv1_relu.forward(out);
        out = conv1_max_pool.forward(out);
        out = layer1_basic0.forward(out);
        out = layer1_basic1.forward(out);
        out = layer2_basic0.forward(out);
        out = layer2_basic1.forward(out);
        out = layer3_basic0.forward(out);
        out = layer3_basic1.forward(out);
        out = layer4_basic0.forward(out);
        out = layer4_basic1.forward(out);
        out = predict.forward(out);
        chkerr(hipDeviceSynchronize());
        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<"resnet18-original,"<<inference_time/1024<<endl;
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,out,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("resnet18-original.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}