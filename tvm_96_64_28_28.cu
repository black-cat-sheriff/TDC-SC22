//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[14];
    __shared__ float pad_temp_shared[448];
    __shared__ float kernel_shared[768];
    float pad_temp_shared_local[56];
    float kernel_shared_local[16];
    for (int yy_c_init = 0; yy_c_init < 7; ++yy_c_init) {
        compute_local[(yy_c_init)] = 0.000000e+00f;
        compute_local[((yy_c_init + 7))] = 0.000000e+00f;
    }
    for (int rc_outer = 0; rc_outer < 12; ++rc_outer) {
        for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
                pad_temp_shared[(((((((int)threadIdx.z) * 28) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 14) + ry_outer) + (((((int)threadIdx.z) * 7) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 2)) % 14))) && ((((((int)blockIdx.y) * 14) + ry_outer) + (((((int)threadIdx.z) * 7) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 2)) % 14)) < 29)) && (1 <= ((((int)blockIdx.x) * 2) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)))) && (((((int)blockIdx.x) * 2) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)) < 29)) ? data[(((((((((rc_outer * 6272) + ((((((int)threadIdx.z) * 7) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 2)) / 14) * 784)) + (((int)blockIdx.y) * 392)) + (ry_outer * 28)) + ((((((int)threadIdx.z) * 7) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 2)) % 14) * 28)) + (((int)blockIdx.x) * 2)) + ((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 3)) - 29))] : 0.000000e+00f);
            }
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
                kernel_shared[(((((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) + (((int)threadIdx.x) * 12)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 1728)) + (((int)threadIdx.y) * 864)) + (rc_outer * 72)) + (((int)threadIdx.x) * 36)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3) * 9)) + (ry_outer * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
            }
            __syncthreads();
            for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
                for (int ax1 = 0; ax1 < 8; ++ax1) {
                    for (int ax2 = 0; ax2 < 7; ++ax2) {
                        pad_temp_shared_local[(((ax1 * 7) + ax2))] = pad_temp_shared[((((((ax1 * 56) + (((int)threadIdx.y) * 28)) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer))];
                    }
                }
                for (int ax11 = 0; ax11 < 8; ++ax11) {
                    kernel_shared_local[(ax11)] = kernel_shared[((((((int)threadIdx.z) * 24) + (ax11 * 3)) + rx_inner_outer))];
                    kernel_shared_local[((ax11 + 8))] = kernel_shared[(((((((int)threadIdx.z) * 24) + (ax11 * 3)) + rx_inner_outer) + 384))];
                }
                for (int rc_inner_inner = 0; rc_inner_inner < 8; ++rc_inner_inner) {
                    for (int yy_c = 0; yy_c < 7; ++yy_c) {
                        compute_local[(yy_c)] = (compute_local[(yy_c)] + (pad_temp_shared_local[(((rc_inner_inner * 7) + yy_c))] * kernel_shared_local[(rc_inner_inner)]));
                        compute_local[((yy_c + 7))] = (compute_local[((yy_c + 7))] + (pad_temp_shared_local[(((rc_inner_inner * 7) + yy_c))] * kernel_shared_local[((rc_inner_inner + 8))]));
                    }
                }
            }
        }
    }
    for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 7; ++yy_inner_inner_inner) {
        compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 196)) + (yy_inner_inner_inner * 28)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = compute_local[(yy_inner_inner_inner)];
        compute[(((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 196)) + (yy_inner_inner_inner * 28)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 12544))] = compute_local[((yy_inner_inner_inner + 7))];
    }
}


//grid=(14,2,2),  block=(2,2,16)
int main(int argc, char *argv[]){
    dim3 grid(14,2,2);
    dim3 block(2,2,16);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}