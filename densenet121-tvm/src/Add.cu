#include "../inc/common.h"
void Add::initialize(unsigned int b, unsigned int c, unsigned int h, unsigned int w) {
    B = b;
    C = c;
    H = h;
    W = w;
    checkCUDNN(hipdnnCreate(&addCudnn));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addInputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addInputDescriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
    checkCUDNN(hipdnnCreateTensorDescriptor(&addOutputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(addOutputDescriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/B,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W));
}
float *Add::forward(float *x, float *y) {
    checkCUDNN(hipdnnAddTensor(addCudnn,&alpha,addInputDescriptor,x,&beta,addOutputDescriptor,y));
    return y;
}