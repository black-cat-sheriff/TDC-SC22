//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[7];
    __shared__ float pad_temp_shared[504];
    __shared__ float kernel_shared[2304];
    float pad_temp_shared_local[14];
    float kernel_shared_local[2];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    compute_local[(2)] = 0.000000e+00f;
    compute_local[(3)] = 0.000000e+00f;
    compute_local[(4)] = 0.000000e+00f;
    compute_local[(5)] = 0.000000e+00f;
    compute_local[(6)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
        for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
            __syncthreads();
            pad_temp_shared[((((int)threadIdx.z) * 16))] = (((((1 <= ((((((int)threadIdx.z) * 16) % 21) / 3) + ry_outer)) && (((((((int)threadIdx.z) * 16) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) - 8))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 1))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 1) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 1) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 1) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 1) % 3)) - 8))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 2))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 2) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 2) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 2) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 2) % 3)) - 8))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 3))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 3) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 3) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) - 1))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 4))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 4) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 4) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 4) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 1) % 3)) - 8))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 5))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 5) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 5) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 5) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 2) % 3)) - 8))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 6))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 6) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 6) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) + 6))] : 0.000000e+00f);
            pad_temp_shared[(((((int)threadIdx.z) * 16) + 7))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 7) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 7) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 7) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 1) % 3)) - 8))] : 0.000000e+00f);
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 8))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 8) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 8) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 8) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 2) % 3)) - 8))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 9))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 9) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 9) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) + 13))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 10))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 10) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 10) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 10) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 1) % 3)) - 8))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 11))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 11) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 11) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 11) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 2) % 3)) - 8))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 12))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 12) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 12) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) + 20))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 13))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 13) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 13) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 1) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 13) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 1) % 3)) - 8))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 14))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 14) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 14) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 16) + 2) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + ((((((int)threadIdx.z) * 16) + 14) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 16) + 2) % 3)) - 8))] : 0.000000e+00f);
            }
            if (((int)threadIdx.z) < 31) {
                pad_temp_shared[(((((int)threadIdx.z) * 16) + 15))] = (((((1 <= (((((((int)threadIdx.z) * 16) + 15) % 21) / 3) + ry_outer)) && ((((((((int)threadIdx.z) * 16) + 15) % 21) / 3) + ry_outer) < 8)) && (1 <= (((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)))) && ((((int)blockIdx.x) + ((((int)threadIdx.z) * 16) % 3)) < 8)) ? data[(((((((rc_outer * 1176) + (((((int)threadIdx.z) * 16) / 3) * 7)) + (ry_outer * 7)) + ((int)blockIdx.x)) + ((((int)threadIdx.z) * 16) % 3)) + 27))] : 0.000000e+00f);
            }
            kernel_shared[((((int)threadIdx.z) * 72))] = kernel[(((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 1))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 1))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 2))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 2))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 3))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 9))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 4))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 10))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 5))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 11))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 6))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 18))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 7))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 19))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 8))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 20))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 9))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 27))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 10))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 28))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 11))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 29))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 12))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 36))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 13))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 37))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 14))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 38))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 15))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 45))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 16))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 46))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 17))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 47))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 18))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 54))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 19))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 55))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 20))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 56))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 21))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 63))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 22))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 64))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 23))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 65))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 24))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 72))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 25))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 73))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 26))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 74))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 27))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 81))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 28))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 82))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 29))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 83))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 30))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 90))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 31))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 91))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 32))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 92))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 33))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 99))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 34))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 100))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 35))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 101))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 36))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 108))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 37))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 109))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 38))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 110))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 39))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 117))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 40))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 118))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 41))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 119))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 42))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 126))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 43))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 127))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 44))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 128))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 45))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 135))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 46))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 136))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 47))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 137))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 48))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 144))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 49))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 145))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 50))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 146))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 51))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 153))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 52))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 154))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 53))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 155))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 54))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 162))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 55))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 163))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 56))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 164))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 57))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 171))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 58))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 172))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 59))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 173))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 60))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 180))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 61))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 181))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 62))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 182))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 63))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 189))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 64))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 190))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 65))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 191))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 66))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 198))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 67))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 199))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 68))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 200))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 69))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 207))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 70))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 208))];
            kernel_shared[(((((int)threadIdx.z) * 72) + 71))] = kernel[((((((((int)blockIdx.z) * 55296) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (ry_outer * 3)) + 209))];
            __syncthreads();
            for (int rc_inner_outer = 0; rc_inner_outer < 12; ++rc_inner_outer) {
                pad_temp_shared_local[(0)] = pad_temp_shared[((rc_inner_outer * 42))];
                pad_temp_shared_local[(1)] = pad_temp_shared[(((rc_inner_outer * 42) + 3))];
                pad_temp_shared_local[(2)] = pad_temp_shared[(((rc_inner_outer * 42) + 6))];
                pad_temp_shared_local[(3)] = pad_temp_shared[(((rc_inner_outer * 42) + 9))];
                pad_temp_shared_local[(4)] = pad_temp_shared[(((rc_inner_outer * 42) + 12))];
                pad_temp_shared_local[(5)] = pad_temp_shared[(((rc_inner_outer * 42) + 15))];
                pad_temp_shared_local[(6)] = pad_temp_shared[(((rc_inner_outer * 42) + 18))];
                pad_temp_shared_local[(7)] = pad_temp_shared[(((rc_inner_outer * 42) + 21))];
                pad_temp_shared_local[(8)] = pad_temp_shared[(((rc_inner_outer * 42) + 24))];
                pad_temp_shared_local[(9)] = pad_temp_shared[(((rc_inner_outer * 42) + 27))];
                pad_temp_shared_local[(10)] = pad_temp_shared[(((rc_inner_outer * 42) + 30))];
                pad_temp_shared_local[(11)] = pad_temp_shared[(((rc_inner_outer * 42) + 33))];
                pad_temp_shared_local[(12)] = pad_temp_shared[(((rc_inner_outer * 42) + 36))];
                pad_temp_shared_local[(13)] = pad_temp_shared[(((rc_inner_outer * 42) + 39))];
                kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)))];
                kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)) + 3))];
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
                pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 42) + 1))];
                pad_temp_shared_local[(1)] = pad_temp_shared[(((rc_inner_outer * 42) + 4))];
                pad_temp_shared_local[(2)] = pad_temp_shared[(((rc_inner_outer * 42) + 7))];
                pad_temp_shared_local[(3)] = pad_temp_shared[(((rc_inner_outer * 42) + 10))];
                pad_temp_shared_local[(4)] = pad_temp_shared[(((rc_inner_outer * 42) + 13))];
                pad_temp_shared_local[(5)] = pad_temp_shared[(((rc_inner_outer * 42) + 16))];
                pad_temp_shared_local[(6)] = pad_temp_shared[(((rc_inner_outer * 42) + 19))];
                pad_temp_shared_local[(7)] = pad_temp_shared[(((rc_inner_outer * 42) + 22))];
                pad_temp_shared_local[(8)] = pad_temp_shared[(((rc_inner_outer * 42) + 25))];
                pad_temp_shared_local[(9)] = pad_temp_shared[(((rc_inner_outer * 42) + 28))];
                pad_temp_shared_local[(10)] = pad_temp_shared[(((rc_inner_outer * 42) + 31))];
                pad_temp_shared_local[(11)] = pad_temp_shared[(((rc_inner_outer * 42) + 34))];
                pad_temp_shared_local[(12)] = pad_temp_shared[(((rc_inner_outer * 42) + 37))];
                pad_temp_shared_local[(13)] = pad_temp_shared[(((rc_inner_outer * 42) + 40))];
                kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)) + 1))];
                kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)) + 4))];
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
                pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 42) + 2))];
                pad_temp_shared_local[(1)] = pad_temp_shared[(((rc_inner_outer * 42) + 5))];
                pad_temp_shared_local[(2)] = pad_temp_shared[(((rc_inner_outer * 42) + 8))];
                pad_temp_shared_local[(3)] = pad_temp_shared[(((rc_inner_outer * 42) + 11))];
                pad_temp_shared_local[(4)] = pad_temp_shared[(((rc_inner_outer * 42) + 14))];
                pad_temp_shared_local[(5)] = pad_temp_shared[(((rc_inner_outer * 42) + 17))];
                pad_temp_shared_local[(6)] = pad_temp_shared[(((rc_inner_outer * 42) + 20))];
                pad_temp_shared_local[(7)] = pad_temp_shared[(((rc_inner_outer * 42) + 23))];
                pad_temp_shared_local[(8)] = pad_temp_shared[(((rc_inner_outer * 42) + 26))];
                pad_temp_shared_local[(9)] = pad_temp_shared[(((rc_inner_outer * 42) + 29))];
                pad_temp_shared_local[(10)] = pad_temp_shared[(((rc_inner_outer * 42) + 32))];
                pad_temp_shared_local[(11)] = pad_temp_shared[(((rc_inner_outer * 42) + 35))];
                pad_temp_shared_local[(12)] = pad_temp_shared[(((rc_inner_outer * 42) + 38))];
                pad_temp_shared_local[(13)] = pad_temp_shared[(((rc_inner_outer * 42) + 41))];
                kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)) + 2))];
                kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 72) + (rc_inner_outer * 6)) + 5))];
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
                compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
                compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(1)]));
                compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(1)]));
                compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
                compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(1)]));
                compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(1)]));
                compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
            }
        }
    }
    compute[((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)))] = compute_local[(0)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 7))] = compute_local[(1)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 14))] = compute_local[(2)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 21))] = compute_local[(3)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 28))] = compute_local[(4)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 35))] = compute_local[(5)];
    compute[(((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 49)) + ((int)blockIdx.x)) + 42))] = compute_local[(6)];
}

//grid=(7,1,5),  block=(1,1,32)
int main(int argc, char *argv[]){
    dim3 grid(7,1,5);
    dim3 block(1,1,32);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}