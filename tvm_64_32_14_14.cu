//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[7];
    __shared__ float pad_temp_shared[1152];
    __shared__ float kernel_shared[1152];
    float pad_temp_shared_local[672];
    float kernel_shared_local[96];
    compute_local[(0)] = 0.000000e+00f;
    compute_local[(1)] = 0.000000e+00f;
    compute_local[(2)] = 0.000000e+00f;
    compute_local[(3)] = 0.000000e+00f;
    compute_local[(4)] = 0.000000e+00f;
    compute_local[(5)] = 0.000000e+00f;
    compute_local[(6)] = 0.000000e+00f;
    for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 144; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
            pad_temp_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 144)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 7) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 36) >> 2))) && (((((int)blockIdx.y) * 7) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 36) >> 2)) < 15)) && (1 <= ((((int)blockIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)))) && (((((int)blockIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)) < 15)) ? data[((((((((((rc_outer * 6272) + (((int)threadIdx.z) * 1568)) + (((int)threadIdx.x) * 784)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 36) * 196)) + (((int)blockIdx.y) * 98)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 36) >> 2) * 14)) + (((int)blockIdx.x) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)) - 15))] : 0.000000e+00f);
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 144; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
            kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 144)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 144)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
        }
        __syncthreads();
        for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
#pragma unroll
            for (int ax1 = 0; ax1 < 32; ++ax1) {
#pragma unroll
                for (int ax2 = 0; ax2 < 3; ++ax2) {
                    pad_temp_shared_local[(((ax1 * 3) + ax2))] = pad_temp_shared[(((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 96))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 4))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 192))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 8))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 288))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 12))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 384))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 16))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 480))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 20))];
                    pad_temp_shared_local[((((ax1 * 3) + ax2) + 576))] = pad_temp_shared[((((((ax1 * 36) + (ax2 * 4)) + ((int)threadIdx.x)) + rx_inner_outer) + 24))];
                }
            }
#pragma unroll
            for (int ax11 = 0; ax11 < 32; ++ax11) {
#pragma unroll
                for (int ax21 = 0; ax21 < 3; ++ax21) {
                    kernel_shared_local[(((ax11 * 3) + ax21))] = kernel_shared[(((((((int)threadIdx.z) * 288) + (ax11 * 9)) + (ax21 * 3)) + rx_inner_outer))];
                }
            }
#pragma unroll
            for (int rc_inner_inner = 0; rc_inner_inner < 32; ++rc_inner_inner) {
#pragma unroll
                for (int ry_inner_inner = 0; ry_inner_inner < 3; ++ry_inner_inner) {
                    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 96))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 192))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 288))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 384))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 480))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[((((rc_inner_inner * 3) + ry_inner_inner) + 576))] * kernel_shared_local[(((rc_inner_inner * 3) + ry_inner_inner))]));
                }
            }
        }
    }
    compute[((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = compute_local[(0)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 14))] = compute_local[(1)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 28))] = compute_local[(2)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 42))] = compute_local[(3)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 56))] = compute_local[(4)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 70))] = compute_local[(5)];
    compute[(((((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 84))] = compute_local[(6)];
}


//grid=(7,2,8),  block=(2,1,4)
int main(int argc, char *argv[]){
    dim3 grid(7,2,8);
    dim3 block(2,1,4);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}