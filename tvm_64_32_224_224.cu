//
// Created by lizhi on 3/24/22.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
using namespace std;
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
    float compute_local[512];
    __shared__ float pad_temp_shared[4352];
    __shared__ float kernel_shared[1536];
    float pad_temp_shared_local[512];
    float kernel_shared_local[256];
    for (int ff_c_init = 0; ff_c_init < 16; ++ff_c_init) {
        for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
            for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
                compute_local[((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 64))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 128))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 192))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 256))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 320))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 384))] = 0.000000e+00f;
                compute_local[(((((ff_c_init * 4) + (yy_c_init * 2)) + xx_c_init) + 448))] = 0.000000e+00f;
            }
        }
    }
    for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
        for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
            __syncthreads();
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 272; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
                pad_temp_shared[(((((((int)threadIdx.z) * 2176) + (((int)threadIdx.y) * 544)) + (((int)threadIdx.x) * 272)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 34)) + ry_outer)) && ((((((int)blockIdx.y) * 8) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 34)) + ry_outer) < 225)) && (1 <= ((((int)blockIdx.x) * 32) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 34)))) && (((((int)blockIdx.x) * 32) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 34)) < 225)) ? data[(((((((((((rc_outer * 802816) + (((int)threadIdx.z) * 401408)) + (((int)threadIdx.y) * 100352)) + (((int)threadIdx.x) * 50176)) + (((int)blockIdx.y) * 1792)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 34) * 224)) + (ry_outer * 224)) + (((int)blockIdx.x) * 32)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 34)) - 225))] : 0.000000e+00f);
            }
            for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 96; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
                kernel_shared[(((((((int)threadIdx.z) * 768) + (((int)threadIdx.y) * 192)) + (((int)threadIdx.x) * 96)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((((int)threadIdx.z) * 9216) + (((int)threadIdx.y) * 2304)) + (((int)threadIdx.x) * 1152)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 48) * 576)) + (rc_outer * 144)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 48) / 3) * 9)) + (ry_outer * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
            }
            __syncthreads();
            for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
                for (int ax1 = 0; ax1 < 16; ++ax1) {
                    for (int ax2 = 0; ax2 < 2; ++ax2) {
                        for (int ax3 = 0; ax3 < 2; ++ax3) {
                            pad_temp_shared_local[((((ax1 * 4) + (ax2 * 2)) + ax3))] = pad_temp_shared[(((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 64))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 4))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 128))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 8))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 192))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 12))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 256))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 16))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 320))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 20))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 384))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 24))];
                            pad_temp_shared_local[(((((ax1 * 4) + (ax2 * 2)) + ax3) + 448))] = pad_temp_shared[((((((((ax1 * 272) + (((int)threadIdx.y) * 68)) + (ax2 * 34)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 28))];
                        }
                    }
                }
                for (int ax0 = 0; ax0 < 16; ++ax0) {
                    for (int ax11 = 0; ax11 < 16; ++ax11) {
                        kernel_shared_local[(((ax0 * 16) + ax11))] = kernel_shared[(((((((int)threadIdx.z) * 768) + (ax0 * 48)) + (ax11 * 3)) + rx_inner_outer))];
                    }
                }
                for (int rc_inner_inner = 0; rc_inner_inner < 16; ++rc_inner_inner) {
                    for (int ff_c = 0; ff_c < 16; ++ff_c) {
                        for (int yy_c = 0; yy_c < 2; ++yy_c) {
                            for (int xx_c = 0; xx_c < 2; ++xx_c) {
                                compute_local[((((ff_c * 4) + (yy_c * 2)) + xx_c))] = (compute_local[((((ff_c * 4) + (yy_c * 2)) + xx_c))] + (pad_temp_shared_local[((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 64))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 64))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 64))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 128))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 128))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 128))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 192))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 192))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 192))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 256))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 256))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 256))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 320))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 320))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 320))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 384))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 384))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 384))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                                compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 448))] = (compute_local[(((((ff_c * 4) + (yy_c * 2)) + xx_c) + 448))] + (pad_temp_shared_local[(((((rc_inner_inner * 4) + (yy_c * 2)) + xx_c) + 448))] * kernel_shared_local[(((ff_c * 16) + rc_inner_inner))]));
                            }
                        }
                    }
                }
            }
        }
    }
    for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 16; ++ff_inner_inner_inner) {
        for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
            for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
                compute[(((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner))] = compute_local[((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 4))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 64))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 8))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 128))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 12))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 192))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 16))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 256))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 20))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 320))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 24))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 384))];
                compute[((((((((((((int)threadIdx.z) * 802816) + (ff_inner_inner_inner * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 448)) + (yy_inner_inner_inner * 224)) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 28))] = compute_local[(((((ff_inner_inner_inner * 4) + (yy_inner_inner_inner * 2)) + xx_inner_inner_inner) + 448))];
            }
        }
    }
}

//grid=(7,28,1),  block=(2,4,2)
int main(int argc, char *argv[]){
    dim3 grid(7,28,1);
    dim3 block(2,4,2);
    int C = atoi(argv[1]);
    int H = atoi(argv[2]);
    int W = atoi(argv[3]);
    int N = atoi(argv[4]);
    int data_size = C*H*W;

    float *input;
    hipMalloc(&input,data_size*sizeof(float));
    float *weight;
    hipMalloc(&weight,N*C*9*sizeof(float));
    float *output;
    hipMalloc(&output,N*H*W*sizeof(float));

    hipDeviceSynchronize();
    float inference_time = 0.0f;
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    for(int i=0;i<1024;++i){
        hipEventRecord(event_start);
        default_function_kernel0<<<grid,block>>>(input,weight,output);
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    cout<<C<<","<<N<<","<<H<<","<<W<<","<<inference_time/1024<<endl;
    return 0;
}