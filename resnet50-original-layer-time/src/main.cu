#include "../inc/common.h"
unsigned int *conv_index = new unsigned int[1];
float *conv_time = new float[100];
int main(int argc,char *argv[]){
    conv_index[0] = 0;
    for(int i=0;i<100;++i){
        conv_time[i] = 0.0f;
    }
    float *dInput;
    hipMalloc(&dInput,224*224*3*sizeof(float));
    Conv conv1_conv;
    conv1_conv.initialize(1,3,224,224,64,3,7,7,
                          2," ", false);
    BatchNorm conv1_bn;
    conv1_bn.initialize(1,64,112,112," ");
    Activation conv1_relu;
    conv1_relu.initialize(1,64,112,112);
    Pool conv1_max_pool;
    conv1_max_pool.initialize(1,64,112,112,1,3,3,HIPDNN_POOLING_MAX,2);

    /*
     * layer 1
     */
    BasicBlock_Downsample layer1_basic0(64,64,64,256,256,
                                        56,56,1);
    BasicBlock layer1_basic1(256,64,64,256,56,56);
    BasicBlock layer1_basic2(256,64,64,256,56,56);
    /*
     * layer 2
     */
    BasicBlock_Downsample layer2_basic0(256,128,128,512,512,
                                        56,56,2);
    BasicBlock layer2_basic1(512,128,128,512,28,28);
    BasicBlock layer2_basic2(512,128,128,512,28,28);
    BasicBlock layer2_basic3(512,128,128,512,28,28);
    /*
     * layer 3
     */
    BasicBlock_Downsample layer3_basic0(512,256,256,1024,1024,
                                        28,28,2);
    BasicBlock layer3_basic1(1024,256,256,1024,14,14);
    BasicBlock layer3_basic2(1024,256,256,1024,14,14);
    BasicBlock layer3_basic3(1024,256,256,1024,14,14);
    BasicBlock layer3_basic4(1024,256,256,1024,14,14);
    BasicBlock layer3_basic5(1024,256,256,1024,14,14);
    /*
     * layer 4
     */
    BasicBlock_Downsample layer4_basic0(1024,512,512,2048,2048,
                                        14,14,2);
    BasicBlock layer4_basic1(2048,512,512,2048,7,7);
    BasicBlock layer4_basic2(2048,512,512,2048,7,7);

    Pool avg_pool;
    avg_pool.initialize(1,2048,7,7,0,7,7,HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,1);
    Conv predict;
    predict.initialize(1,2048,1,1,1000,0,1,1,1," ", true);

    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    //hipMemcpy(dInput, input, 224 * 224 * 3 * sizeof(float), hipMemcpyHostToDevice);
    float *out;
    out = conv1_conv.forward(dInput);
    out = conv1_bn.forward(out);
    out = conv1_relu.forward(out);
    out = conv1_max_pool.forward(out);
    out = layer1_basic0.forward(out);
    out = layer1_basic1.forward(out);
    out = layer1_basic2.forward(out);
    out = layer2_basic0.forward(out);
    out = layer2_basic1.forward(out);
    out = layer2_basic2.forward(out);
    out = layer2_basic3.forward(out);

    out = layer3_basic0.forward(out);
    out = layer3_basic1.forward(out);
    out = layer3_basic2.forward(out);
    out = layer3_basic3.forward(out);
    out = layer3_basic4.forward(out);
    out = layer3_basic5.forward(out);

    out = layer4_basic0.forward(out);
    out = layer4_basic1.forward(out);
    out = layer4_basic2.forward(out);

    out = predict.forward(out);
    hipDeviceSynchronize();
    cout<<"network construction finished"<<endl;
    float inference_time = 0.0f;
    for(int i=0;i<100;++i){
        conv_time[i] = 0.0f;
    }
    for(int i=0;i<1024;++i){
        conv_index[0] = 0;
        hipEventRecord(event_start);
        out = conv1_conv.forward(dInput);
        out = conv1_bn.forward(out);
        out = conv1_relu.forward(out);
        out = conv1_max_pool.forward(out);
        out = layer1_basic0.forward(out);
        out = layer1_basic1.forward(out);
        out = layer1_basic2.forward(out);
        out = layer2_basic0.forward(out);
        out = layer2_basic1.forward(out);
        out = layer2_basic2.forward(out);
        out = layer2_basic3.forward(out);

        out = layer3_basic0.forward(out);
        out = layer3_basic1.forward(out);
        out = layer3_basic2.forward(out);
        out = layer3_basic3.forward(out);
        out = layer3_basic4.forward(out);
        out = layer3_basic5.forward(out);

        out = layer4_basic0.forward(out);
        out = layer4_basic1.forward(out);
        out = layer4_basic2.forward(out);

        out = predict.forward(out);

        hipDeviceSynchronize();
        hipEventRecord(event_stop);
        hipEventSynchronize(event_stop);
        float temp_time;
        hipEventElapsedTime(&temp_time, event_start, event_stop);
        inference_time += temp_time;
    }
    for(int i=0;i<100;++i){
        cout<<conv_time[i]/1024<<endl;
    }
    unsigned int outputSize = 1000;
    float *hOutput = (float *)malloc(outputSize*sizeof(float));
    hipMemcpy(hOutput,out,outputSize*sizeof(float),hipMemcpyDeviceToHost);
    vector<float> out_bin;
    for(int i=0;i<outputSize;++i){
        out_bin.push_back(hOutput[i]);
    }
    std::ofstream ofp("resnet50-original.bin", std::ios::out | std::ios::binary);
    ofp.write(reinterpret_cast<const char*>(out_bin.data()), out_bin.size() * sizeof(float));
    return 0;
}